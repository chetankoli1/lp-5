#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function for vector addition
__global__ void vectorAddition(const int* a, const int* b, int* result, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

// Kernel function for matrix multiplication
__global__ void matrixMultiplication(const int* a, const int* b, int* result, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += a[row * size + i] * b[i * size + col];
        }
        result[row * size + col] = sum;
    }
}

int main() {
    int size = 1024; // Size of the vectors and matrices

    // Allocate memory on the host (CPU)
    int* host_a = new int[size];
    int* host_b = new int[size];
    int* host_result = new int[size];

    // Initialize the vectors and matrices
    for (int i = 0; i < size; i++) {
        host_a[i] = i;
        host_b[i] = i;
    }

    // Allocate memory on the device (GPU)
    int* device_a;
    int* device_b;
    int* device_result;

    hipMalloc((void**)&device_a, size * sizeof(int));
    hipMalloc((void**)&device_b, size * sizeof(int));
    hipMalloc((void**)&device_result, size * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(device_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, host_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Vector addition
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    vectorAddition<<<gridSize, blockSize>>>(device_a, device_b, device_result, size);

    // Copy result back from device to host
    hipMemcpy(host_result, device_result, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result of vector addition
    std::cout << "Vector Addition Result:" << std::endl;
    for (int i = 0; i < size; i++) {
        std::cout << host_result[i] << " ";
    }
    std::cout << std::endl;

    // Matrix multiplication
    dim3 blockDim(16, 16);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x, (size + blockDim.y - 1) / blockDim.y);

    matrixMultiplication<<<gridDim, blockDim>>>(device_a, device_b, device_result, size);

    // Copy result back from device to host
    hipMemcpy(host_result, device_result, size * size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result of matrix multiplication
    std::cout << "Matrix Multiplication Result:" << std::endl;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            std::cout << host_result[i * size + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_result);

    // Free host memory
    delete[] host_a;
    delete[] host_b;
    delete[] host_result;

    return 0;
}
